#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "picojson.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

const int THREADS = 8;
const int WORD_SIZE = 10;
const int RESULT_SIZE = WORD_SIZE * 2;

int* intArr;
double* doubleArr;
char* charArr;
char* resultsArr;
int count = 0;

void Read() {
	std::cout << "Pasirinkit failą: ";
	int fileNr;
	std::cin >> fileNr;
	std::string FILENAME = "IFK9-2_GrosasL_L3_dat_";

	switch (fileNr) {
		case (2):
			FILENAME += "2.json";
			break;
		case(3):
			FILENAME += "3.json";
			break;
		default:
			FILENAME += "1.json";
			break;
	}

	std::cout << FILENAME << std::endl;
	std::ifstream dataFile(FILENAME);

	if (!dataFile) {
		std::cout << "Neatidare";
	}
	
	std::string json;
	std::string line;
	json = "";

	while (dataFile >> line) {
		json += line;
	}

	picojson::value v;
	picojson::parse(v, json);
	picojson::array arr = v.get("players").get<picojson::array>();
	count = arr.capacity();

	intArr = new int[count];
	doubleArr = new double[count];
	charArr = (char*)calloc(count * WORD_SIZE, sizeof(char));
	resultsArr = (char*)calloc(count * RESULT_SIZE, sizeof(char));
	int index = 0;

	for (picojson::array::iterator iter = arr.begin(); iter != arr.end(); ++iter) {
		double intArrDbl = (*iter).get("pts").get<double>();
		double dbl = (*iter).get("fgPct").get<double>();
		std::string name = (*iter).get("name").get<std::string>();
		char *carr = new char[name.length()];
		std::strcpy(carr, name.c_str());

		for (int i = 0; i < WORD_SIZE; i++) {
			if (i < name.length()) {
				charArr[WORD_SIZE * index + i] = carr[i];
			}
			else {
				break;
			}
		}

		intArr[index] = (int)intArrDbl;
		doubleArr[index] = dbl;
		index++;
	}
}


__global__ void filterGPU(int* d_int, double* d_double, char* d_chars, int* startIndices, int* endIndices, char* d_results, int *d_count, int *d_resultCount) {
	char* word = new char[WORD_SIZE];
	int threadID = threadIdx.x;
	int nameLength;
	char letter;

	for (int j = startIndices[threadID]; j < endIndices[threadID]; j++) {
		word = new char[WORD_SIZE];
		nameLength = 0;
		for (int i = 0; i < WORD_SIZE; i++) {
			letter = d_chars[j * WORD_SIZE + i];
			if (letter != NULL) {
				word[i] = letter;
				nameLength++;
			}
			else { break; }
		}

		char* result = new char[RESULT_SIZE];
		double compValue = d_double[j] * d_int[j];
		int whole = (int)compValue;
		
		if (compValue - (double)whole == 0) {
			char grade;

			if (whole > 4000) { grade = 'A'; }
			else if (whole > 3000) { grade = 'B'; }
			else if (whole > 2000) { grade = 'C'; }
			else if (whole > 1000) { grade = 'D'; }
			else { grade = 'F'; }

			printf("Thread:%d Name: %s Pts: %d Double: %f Comp:%f \n", threadIdx.x, word, d_int[j], d_double[j], compValue);
			for (int i = 0; i < nameLength; i++) {
				result[i] = word[i];
			}
			result[nameLength] = '-';
			result[nameLength + 1] = grade;
			for (int i = nameLength + 2; i < RESULT_SIZE; i++) {
				result[i] = ' ';
			}

			int index = atomicAdd(d_resultCount, 1);
			int position = index * RESULT_SIZE;

			for (int i = position; i < position + RESULT_SIZE; i++) {
				d_results[i] = result[i - position];
			}
		}
	}
}


int main()
{
	int* d_int;
	double* d_double;
	char* d_chars;
	char* d_results;
	int* startIndices = new int[THREADS];
	int* endIndices = new int[THREADS];
	int* d_startIndices;
	int* d_endIndices;
	int* d_count;
	int* d_resultCount;
	int resultCount = 0;

	// Failo nuskaitymas
	Read();
	
	// Duomenu paskirstymas
	int chunkSize = count / THREADS;
	int overFlow = count % THREADS;
	int tempOverFlow = count % THREADS;
	int offSet;
	int startIndex, endIndex;
	for (int i = 0; i < THREADS; i++) {
		if (i < overFlow) {
			offSet = overFlow - tempOverFlow;
			startIndex = i * chunkSize + offSet;
			endIndex = startIndex + chunkSize + 1;
			std::cout << "Thread:" << i << " Start=" << startIndex << " End=" << endIndex << std::endl;
			tempOverFlow--;
			startIndices[i] = startIndex;
			endIndices[i] = endIndex;
		}
		else {
			startIndex = i * chunkSize + overFlow;
			endIndex = startIndex + chunkSize;
			std::cout << "Thread:" << i << " Start=" << startIndex << " End=" << endIndex << std::endl;
			startIndices[i] = startIndex;
			endIndices[i] = endIndex;
		}
	}


	hipMalloc((void**)&d_count, sizeof(int));
	hipMalloc((void**)&d_resultCount, sizeof(int));
	hipMalloc(&d_int, sizeof(int) * count);
	hipMalloc(&d_double, sizeof(double) * count);
	hipMalloc(&d_chars, sizeof(char) * WORD_SIZE * count);
	hipMalloc(&d_results, sizeof(char) * RESULT_SIZE * count);
	hipMalloc(&d_startIndices, sizeof(int) * THREADS);
	hipMalloc(&d_endIndices, sizeof(int) * THREADS);

	hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_resultCount, &resultCount, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_int, intArr, sizeof(int) * count, hipMemcpyHostToDevice);
	hipMemcpy(d_double, doubleArr, sizeof(double) * count, hipMemcpyHostToDevice);
	hipMemcpy(d_chars, charArr, sizeof(char) * WORD_SIZE * count, hipMemcpyHostToDevice);
	hipMemcpy(d_results, resultsArr, sizeof(char) * RESULT_SIZE * count, hipMemcpyHostToDevice);
	hipMemcpy(d_startIndices, startIndices, sizeof(int) * THREADS, hipMemcpyHostToDevice);
	hipMemcpy(d_endIndices, endIndices, sizeof(int) * THREADS, hipMemcpyHostToDevice);

	filterGPU<<< 1, THREADS >>>(d_int, d_double, d_chars, d_startIndices, d_endIndices, d_results, d_count, d_resultCount);

	hipDeviceSynchronize();

	hipMemcpy(resultsArr, d_results, sizeof(char) * count * RESULT_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(&resultCount, d_resultCount, sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < resultCount * RESULT_SIZE; i++) {
		if (i % RESULT_SIZE == 0) {
			std::cout << "\nNR: " << (i/RESULT_SIZE)+1 << " ";
		}
		std::cout << resultsArr[i];
	}

	// Isvalyti atminti
	hipFree(d_int);
	hipFree(d_double);
	hipFree(d_chars);
	hipFree(d_results);
	hipFree(d_startIndices);
	hipFree(d_endIndices);
	hipFree(d_count);
	hipFree(d_resultCount);

	delete[] intArr;
	delete[] doubleArr;
	delete[] charArr;
	delete[] startIndices;
	delete[] endIndices;

	return 0;
}
